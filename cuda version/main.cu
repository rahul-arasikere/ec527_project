#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <time.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define PLATEAU 0
#define BLOCK_SIZE 16

// Convert 2D index to 1D index.
#define INDEX(j, i, ld) ((j)*ld + (i))

// Convert local (shared memory) coord to global (image) coordinate.
#define L2I(ind, off) (((ind) / BLOCK_SIZE) * (BLOCK_SIZE - 2) - 1 + (off))

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans)                           \
    {                                                 \
        gpuAssert((ans), (char *)__FILE__, __LINE__); \
    }

inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__constant__ int neighbour_x[8] = {-1, 0, 1, 1, 1, 0, -1, -1};
__constant__ int neighbour_y[8] = {-1, -1, -1, 0, 1, 1, 1, 0};

typedef unsigned char image_t, *image_ptr_t;
typedef int img_t, *img_ptr_t;

texture<img_t, 2, hipReadModeElementType> image;

img_ptr_t convert2data(image_ptr_t image, const int width, const int height);
image_ptr_t convert2image(img_ptr_t image, const int width, const int height);
__global__ void steepest_descent_kernel(img_ptr_t in_out, const int width, const int height);
__global__ void increment_kernel(img_ptr_t in_out, const int width, const int height);
__global__ void border_kernel(img_ptr_t in_out, int *count, const int width, const int height);
__global__ void minima_basin_kernel(img_ptr_t in_out, int *count, const int width, const int height);
__global__ void watershed_kernel(img_ptr_t in_out, int *count, const int width, const int height);
double interval(struct timespec start, struct timespec end);
int main(int argc, char **argv);

int main(int argc, char **argv)
{
    int width, height, channels;
    image_ptr_t data = stbi_load(argv[1], &width, &height, &channels, 1);
    img_ptr_t input = convert2data(data, width, height);
    stbi_image_free(data);
    img_ptr_t cpu_lowest_descent = (img_ptr_t)calloc(width * height, sizeof(img_t));
    img_ptr_t cpu_border = (img_ptr_t)calloc(width * height, sizeof(img_t));
    img_ptr_t cpu_minima = (img_ptr_t)calloc(width * height, sizeof(img_t));
    img_ptr_t cpu_watershed = (img_ptr_t)calloc(width * height, sizeof(img_t));
    if (cpu_border == NULL || cpu_lowest_descent == NULL || cpu_minima == NULL || cpu_watershed == NULL)
    {
        fprintf(stderr, "Failed to allocate memory!\n");
        exit(EXIT_FAILURE);
    }
    img_ptr_t gpu_memory;
    size_t offset = 0;
    CUDA_SAFE_CALL(hipSetDevice(0));
    CUDA_SAFE_CALL(hipMalloc((img_ptr_t *)&gpu_memory, width * height * sizeof(img_t)));
    CUDA_SAFE_CALL(hipMalloc((img_ptr_t *)&image, width * height * sizeof(img_t)));
    CUDA_SAFE_CALL(hipMemcpy(gpu_memory, input, width * height * sizeof(img_t), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipBindTexture(&offset, image, gpu_memory, width * height * sizeof(img_t)));
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 blocksPerGrid(width / (threadsPerBlock.x - 2), height / (threadsPerBlock.y - 2), 1);
    int *count = NULL;
    int _old = -1;
    int _new = -2;
    CUDA_SAFE_CALL(hipMallocManaged((int **)&count, sizeof(int)));
    steepest_descent_kernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_memory, width, height);
    CUDA_SAFE_CALL(hipPeekAtLastError());
    CUDA_SAFE_CALL(hipMemcpy(cpu_lowest_descent, gpu_memory, width * height * sizeof(img_t), hipMemcpyDeviceToHost));
    increment_kernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_memory, width, height);
    CUDA_SAFE_CALL(hipPeekAtLastError());
    *count = 0;
    while (_old != _new)
    {
        _old = _new;
        border_kernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_memory, count, width, height);
        CUDA_SAFE_CALL(hipPeekAtLastError());
        _new = *count;
    }
    CUDA_SAFE_CALL(hipMemcpy(cpu_border, gpu_memory, width * height * sizeof(img_t), hipMemcpyDeviceToHost));
    *count = 0;
    _old = -1;
    _new = -2;
    while (_old != _new)
    {
        _old = _new;
        minima_basin_kernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_memory, count, width, height);
        CUDA_SAFE_CALL(hipPeekAtLastError());
        _new = *count;
    }
    CUDA_SAFE_CALL(hipMemcpy(cpu_minima, gpu_memory, width * height * sizeof(img_t), hipMemcpyDeviceToHost));
    *count = 0;
    _old = -1;
    _new = -2;
    while (_old != _new)
    {
        _old = _new;
        watershed_kernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_memory, count, width, height);
        CUDA_SAFE_CALL(hipPeekAtLastError());
        _new = *count;
    }
    CUDA_SAFE_CALL(hipMemcpy(cpu_watershed, gpu_memory, width * height * sizeof(img_t), hipMemcpyDeviceToHost));
    stbi_write_png("1_lowest_descent_result.png", width, height, channels, convert2image(cpu_lowest_descent, width, height), width * channels);
    stbi_write_png("2_border_result.png", width, height, channels, convert2image(cpu_border, width, height), width * channels);
    stbi_write_png("3_minima_basin_result.png", width, height, channels, convert2image(cpu_minima, width, height), width * channels);
    stbi_write_png("4_watershed_result.png", width, height, channels, convert2image(cpu_watershed, width, height), width * channels);
    CUDA_SAFE_CALL(hipDeviceReset());
    return 0;
}

double interval(struct timespec start, struct timespec end)
{
    /*
    This method does not require adjusting a #define constant

    How to use this method:

        struct timespec time_start, time_stop;
        clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_start);
        // DO SOMETHING THAT TAKES TIME
        clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_stop);
        measurement = interval(time_start, time_stop);*/
    struct timespec temp;
    temp.tv_sec = end.tv_sec - start.tv_sec;
    temp.tv_nsec = end.tv_nsec - start.tv_nsec;
    if (temp.tv_nsec < 0)
    {
        temp.tv_sec = temp.tv_sec - 1;
        temp.tv_nsec = temp.tv_nsec + 1000000000;
    }
    return (((double)temp.tv_sec) + ((double)temp.tv_nsec) * 1.0e-9);
}

img_ptr_t convert2data(image_ptr_t image, const int width, const int height)
{
    img_ptr_t temp = (img_ptr_t)calloc(width * height, sizeof(img_t));
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            temp[i * width + j] = (img_t)image[i * width + j];
        }
    }
    return temp;
}

image_ptr_t convert2image(img_ptr_t image, const int width, const int height)
{
    // Step 1: find min and max values from the image
    img_t max = INT_MIN, min = INT_MAX;
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width; j++)
        {
            img_t current_pixel = image[i * width + j];
            if (current_pixel < min)
                min = current_pixel;
            if (current_pixel > max)
                max = current_pixel;
        }

    // Step 2: create a new image with the values scaled from [0-255]
    image_ptr_t temp = (image_ptr_t)calloc(width * height, sizeof(image_t));
    float max_min = max - min;
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            img_t pix_val = image[i * width + j];
            float val = (pix_val - min) / max_min;
            temp[i * width + j] = (image_t)(val * 255);
        }
    }
    return temp;
}

__global__ void increment_kernel(img_ptr_t in_out, const int width, const int height)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int p = INDEX(j, i, width);

    if (j < height && i < width && in_out[p] == PLATEAU)
    {
        in_out[p] += 1;
    }
}

__global__ void steepest_descent_kernel(img_ptr_t in_out, const int width, const int height)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bdx = blockDim.x;
    int bdy = blockDim.y;
    int i = bdx * bx + tx;
    int j = bdy * by + ty;

    __shared__ float s_I[BLOCK_SIZE * BLOCK_SIZE];
    int size = BLOCK_SIZE - 2;
    int img_x = L2I(i, tx);
    int img_y = L2I(j, ty);
    int new_w = width + width * 2;
    int new_h = height + height * 2;
    int p = INDEX(img_y, img_x, width);

    int ghost = (tx == 0 || ty == 0 ||
                 tx == bdx - 1 || ty == bdy - 1);

    if ((bx == 0 && tx == 0) || (by == 0 && ty == 0) ||
        (bx == (width / size - 1) && tx == bdx - 1) ||
        (by == (height / size - 1) && ty == bdy - 1))
    {
        s_I[INDEX(ty, tx, BLOCK_SIZE)] = INFINITY;
    }
    else
    {
        s_I[INDEX(ty, tx, BLOCK_SIZE)] = tex2D(image, img_x, img_y);
    }

    __syncthreads();

    if (j < new_h && i < new_w && ghost == 0)
    {
        float I_q_min = INFINITY;
        float I_p = tex2D(image, img_x, img_y);

        int exists_q = 0;

        for (int k = 0; k < 8; k++)
        {
            int n_x = neighbour_x[k] + tx;
            int n_y = neighbour_y[k] + ty;
            float I_q = s_I[INDEX(n_y, n_x, BLOCK_SIZE)];
            if (I_q < I_q_min)
                I_q_min = I_q;
        }

        for (int k = 0; k < 8; k++)
        {
            int x = neighbour_x[k];
            int y = neighbour_y[k];
            int n_x = x + tx;
            int n_y = y + ty;
            int n_tx = L2I(i, n_x);
            int n_ty = L2I(j, n_y);
            float I_q = s_I[INDEX(n_y, n_x, BLOCK_SIZE)];
            int q = INDEX(n_ty, n_tx, width);
            if (I_q < I_p && I_q == I_q_min)
            {
                in_out[p] = -q;
                exists_q = 1;
                break;
            }
        }
        if (exists_q == 0)
            in_out[p] = PLATEAU;
    }
}

__global__ void border_kernel(img_ptr_t in_out, int *count, const int width, const int height)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bdx = blockDim.x;
    int bdy = blockDim.y;
    int i = bdx * bx + tx;
    int j = bdy * by + ty;

    __shared__ float s_L[BLOCK_SIZE * BLOCK_SIZE];
    int size = BLOCK_SIZE - 2;
    int img_x = L2I(i, tx);
    int img_y = L2I(j, ty);
    int true_p = INDEX(img_y, img_x, width);
    int s_p = INDEX(ty, tx, BLOCK_SIZE);
    int new_w = width + width * 2;
    int new_h = height + height * 2;
    int ghost = (tx == 0 || ty == 0 ||
                 tx == bdx - 1 || ty == bdy - 1)
                    ? 1
                    : 0;

    if ((bx == 0 && tx == 0) || (by == 0 && ty == 0) ||
        (bx == (width / size - 1) && tx == bdx - 1) ||
        (by == (height / size - 1) && ty == bdy - 1))
    {
        s_L[INDEX(ty, tx, BLOCK_SIZE)] = INFINITY;
    }
    else
    {
        s_L[s_p] = in_out[INDEX(img_y, img_x, width)];
    }

    __syncthreads();

    int active = (j < new_h && i < new_w && s_L[s_p] > 0) ? 1 : 0;

    if (active == 1 && ghost == 0)
    {
        for (int k = 0; k < 8; k++)
        {
            int n_x = neighbour_x[k] + tx;
            int n_y = neighbour_y[k] + ty;
            int s_q = INDEX(n_y, n_x, BLOCK_SIZE);
            if (s_L[s_q] == INFINITY)
                continue;
            if (s_L[s_q] > s_L[s_p])
                s_L[s_p] = s_L[s_q];
        }
        if (in_out[true_p] != s_L[s_p])
        {
            in_out[true_p] = s_L[s_p];
            atomicAdd(count, 1);
        }
    }
}

__global__ void minima_basin_kernel(img_ptr_t in_out, int *count, const int width, const int height)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bdx = blockDim.x;
    int bdy = blockDim.y;
    int i = bdx * bx + tx;
    int j = bdy * by + ty;

    __shared__ float s_L[BLOCK_SIZE * BLOCK_SIZE];
    int size = BLOCK_SIZE - 2;
    int img_x = L2I(i, tx);
    int img_y = L2I(j, ty);
    int true_p = INDEX(img_y, img_x, width);
    int p = INDEX(ty, tx, BLOCK_SIZE);
    int new_w = width + width * 2;
    int new_h = height + height * 2;
    int ghost = (tx == 0 || ty == 0 ||
                 tx == bdx - 1 || ty == bdy - 1);

    // Load data into shared memory.
    if ((bx == 0 && tx == 0) || (by == 0 && ty == 0) ||
        (bx == (width / size - 1) && tx == bdx - 1) ||
        (by == (height / size - 1) && ty == bdy - 1))
    {
        s_L[INDEX(ty, tx, BLOCK_SIZE)] = INFINITY;
    }
    else
    {
        s_L[INDEX(ty, tx, BLOCK_SIZE)] = in_out[INDEX(img_y, img_x, width)];
    }

    __syncthreads();

    if (j < new_h && i < new_w &&
        s_L[p] == PLATEAU && ghost == 0)
    {
        float I_p = tex2D(image, img_x, img_y);
        float I_q;
        int n_x, n_y;
        float L_q;

        for (int k = 0; k < 8; k++)
        {
            n_x = neighbour_x[k] + tx;
            n_y = neighbour_y[k] + ty;
            L_q = s_L[INDEX(n_y, n_x, BLOCK_SIZE)];
            if (L_q == INFINITY || L_q >= 0)
                continue;
            int n_tx = L2I(i, n_x);
            int n_ty = L2I(j, n_y);
            int q = INDEX(n_ty, n_tx, width);
            I_q = tex2D(image, n_tx, n_ty);
            if (I_q == I_p && in_out[true_p] != -q)
            {
                in_out[true_p] = -q;
                atomicAdd(count, 1);
                break;
            }
        }
    }
}

__global__ void watershed_kernel(img_ptr_t in_out, int *count, const int width, const int height)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int p = INDEX(j, i, width);
    int q;

    if (j < height && i < width && in_out[p] <= 0)
    {
        q = -in_out[p];
        if (in_out[q] > 0 && in_out[p] != in_out[q])
        {
            in_out[p] = in_out[q];
            atomicAdd(count, 1);
        }
    }
}